#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {

__global__ void vertical_flip_kernel_shared(const uint8_t* input, uint8_t* output, int width, int height, int channels) {
    extern __shared__ uint8_t tile[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x  = blockIdx.x * blockDim.x + tx;
    int y  = blockIdx.y * blockDim.y + ty;

    int local_idx = (ty * blockDim.x + tx) * channels;

    if (x < width && y < height) {
        // Load pixel into shared memory
        for (int c = 0; c < channels; ++c) {
            tile[local_idx + c] = input[(y * width + x) * channels + c];
        }
    }

    __syncthreads();

    if (x < width && y < height) {
        // Global vertical flip index
        int flipped_y = height - 1 - y;
        for (int c = 0; c < channels; ++c) {
            output[(flipped_y * width + x) * channels + c] = tile[local_idx + c];
        }
    }
}

void vertical_flip(uint8_t* input, uint8_t* output, int width, int height, int channels) {
    uint8_t *d_input, *d_output;
    size_t size = width * height * channels * sizeof(uint8_t);
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1)/block.x,
              (height + block.y - 1)/block.y);

    size_t sharedMemSize = block.x * block.y * channels * sizeof(uint8_t);

    vertical_flip_kernel_shared<<<grid, block, sharedMemSize>>>(d_input, d_output, width, height, channels);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

}