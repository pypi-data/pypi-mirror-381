#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {

// Each pixel is 3 bytes (RGB)
__global__ void vertical_flip_kernel(const uint8_t* input, uint8_t* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        for (int c = 0; c < channels; ++c) {
            output[(y*width + x)*channels + c] = input[((height - 1 - y)*width + x)*channels + c];
        }
    }
}

void vertical_flip(uint8_t* input, uint8_t* output, int width, int height, int channels) {
    uint8_t *d_input, *d_output;
    size_t size = width * height * channels * sizeof(uint8_t);
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    dim3 block(16,16);
    dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);
    vertical_flip_kernel<<<grid, block>>>(d_input, d_output, width, height, channels);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

}
