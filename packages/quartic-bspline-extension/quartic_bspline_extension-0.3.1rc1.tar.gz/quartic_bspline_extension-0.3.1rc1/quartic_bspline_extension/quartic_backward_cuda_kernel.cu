#include "hip/hip_runtime.h"
#include <vector>
#include <torch/extension.h>

#include "include/constants.h"
#include "include/debug_utils.cuh"
#include "include/index_utils.h"
#include "include/device_utils.h"

/**
 * @brief CUDA kernel implementing the backward step of quartic (midpoint cardinal) 
 *      b-spline potential functions. It computes the derivative of the potential
 *      w.r.t. the weights.
 *
 * @note 
 *  > The derivative w.r.t. the state is computed within the kernel for the 
 *    forward step. Aggregation is managed within custom autograd function on 
 *    PyTorch side.
 *  > Possible optimisations in terms of speed:
 *      * Introduce shared memory on CUDA thread block
 *      * Compute partial gradients on thread block and store results into 
 *        shared memory first.
 *      * Write to global memory only after all threads of the block finished
 *        their work.  
 * 
 * @tparam T Floating point type: float, or double.
 * @param x Tensor of shape [bs, f, w, h] at which the b-spline has to be evaluated.
 * @param weight_tensor Tensor of shape [f, num_centers] corresponding to the 
 *      weights spline potential at the center nodes for each marginal.
 * @param centers Tensor of shape [num_centers, ] of center nodes.
 * @param scale Scaling parameter.
 * @param scale_inv Inverse of the scaling parameter.
 * @param delta_inv Inverse of distance between (equally spaced) center nodes.
 * @param grad_out Tensor of shape [bs, f, w, h] corresponding to the gradient 
 *      of (scalar) loss w.r.t. the output of spline potential.
 * @param grad_w Tensor of shape [f, num_centers] of derivatives w.r.t. to 
 *      the weights of the spline potential evaluated at weight_tensor.
 */
template <typename T>
__global__ void quartic_bspline_backward_cuda_kernel(
    const torch::PackedTensorAccessor32<T, 4> x,
    const torch::PackedTensorAccessor32<T, 2> weight_tensor,
    const torch::PackedTensorAccessor32<T, 1> centers,
    const T scale,
    const T scale_inv,
    const T delta_inv,
    const torch::PackedTensorAccessor32<T, 4> grad_out,
    torch::PackedTensorAccessor32<T, 2> grad_w
){
    const int64_t num_centers = centers.size(0);
    const int64_t num_features = x.size(1);

    const int64_t idx_h = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t idx_w = blockIdx.y * blockDim.y + threadIdx.y;
    const int64_t idx_bf = blockIdx.z;

    const int64_t idx_bs = idx_bf / num_features;
    const int64_t idx_f = idx_bf % num_features;

    if (idx_bs < x.size(0) && idx_f < num_features && idx_w < x.size(2) && idx_h < x.size(3)){

        const T x_ = x[idx_bs][idx_f][idx_w][idx_h];

        const std::pair<int, int> center_idx_bounds = 
                    compute_center_index_bounds(x_, centers[0], scale, delta_inv, centers.size(0));

        for (int j = center_idx_bounds.first; j <= center_idx_bounds.second; j++){
            const T x_scaled = (x_ - centers[j]) * scale_inv;
            if (fabsf(x_scaled) < SUPP_RAD){               
                
                // determine support interval
                int interval = static_cast<int>(x_scaled - SUPP_LOWER);
                interval = max(0, min(NUM_SUPP_INTERVALS - 1, interval));
                
                // evaluate local spline
                T spline_val = QUARTIC_BSPLINE_COEFFS[interval][4];
                #pragma unroll
                for (int i = 1; i <= NUM_SUPP_INTERVALS - 1; i++){
                    spline_val = spline_val * x_scaled 
                               + QUARTIC_BSPLINE_COEFFS[interval][NUM_SUPP_INTERVALS - 1 - i];
                }

                atomicAdd(&grad_w[idx_f][j], grad_out[idx_bs][idx_f][idx_w][idx_h] * spline_val);
            }
        }
    }
}

std::vector<torch::Tensor> quartic_bspline_backward_cuda_function(
    const torch::Tensor x,
    const torch::Tensor weight_tensor,
    const torch::Tensor centers,
    const double scale,
    const torch::Tensor grad_out
){
    check_device_and_datatype({x, weight_tensor, centers, grad_out});

    const dim3 block_size(32, 8);
    const dim3 grid_size((x.size(3) + block_size.x - 1) / block_size.x, 
                         (x.size(2) + block_size.y - 1) / block_size.y,
                         x.size(0) * x.size(1));

    // Initialisation with zero is important here!!
    auto grad_w = torch::zeros_like(weight_tensor);

    const double scale_inv = 1.0 / scale;
    const double delta_inv = 1.0 / (centers[1].item<double>() - centers[0].item<double>());

    auto scalar_type = x.scalar_type();
    AT_DISPATCH_FLOATING_TYPES(scalar_type, "quartic_bspline_backward_cuda", [&] {
        quartic_bspline_backward_cuda_kernel<scalar_t><<<grid_size, block_size>>>(
            x.packed_accessor32<scalar_t, 4>(),
            weight_tensor.packed_accessor32<scalar_t, 2>(), 
            centers.packed_accessor32<scalar_t, 1>(),
            static_cast<scalar_t>(scale),
            static_cast<scalar_t>(scale_inv),
            static_cast<scalar_t>(delta_inv),
            grad_out.packed_accessor32<scalar_t, 4>(),
            grad_w.packed_accessor32<scalar_t, 2>()
        );
    });

    CUDA_DEBUG_FUNC(hipGetLastError());

    return {grad_w};
}